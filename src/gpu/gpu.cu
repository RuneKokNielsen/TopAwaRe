#include "hip/hip_runtime.h"
#
#include "gpu.h"
#include <stdio.h>
#include <unistd.h>

void cuda_set_device(int devid){
   hipSetDevice(devid);
   cuda_error_handle();

   hipDeviceReset();
   cuda_error_handle();

   /**
    * To prepare the device, shortly allocate most
    * available memory. There is no mention of this
    * in the cuda docs, but it prevents initialization
    * issues when the device is coming up from sleep.
    **/
   size_t free;
   size_t total;
   hipMemGetInfo(&free, &total);
   char *data;
   
   hipMalloc(&data, free * 0.9);
   cuda_error_handle();

   hipFree(data);
   cuda_error_handle();

   
}

/**
 * Helpers
 **/
__global__
void kernel_zerofill(int n, float *a){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) a[i] = 0;
}
float *cuda_f32_allocate(int n, bool zerofill){
  float *g_a;
  hipMalloc(&g_a, n * sizeof(float));
  hipError_t errSync  = hipGetLastError();
  while(errSync == 2){
    printf("Not enough memory available! Stalling...\n");
    hipMalloc(&g_a, n * sizeof(float));
    errSync  = hipGetLastError();
    usleep(10000000);
  }
  if(zerofill) kernel_zerofill<<<n/100,256>>>(n, g_a);
  cuda_error_handle();
  return g_a;
}

void cuda_f32_free(float *g_a){
  hipFree(g_a);
  cuda_error_handle();
}

void cuda_f32_send(int n, float *a, float *g_a){
  hipMemcpy(g_a, a, n*sizeof(float), hipMemcpyHostToDevice);
  cuda_error_handle();
}

void cuda_f32_retrieve(int n, float *a, float *g_a){
  hipMemcpy(a, g_a, n*sizeof(float), hipMemcpyDeviceToHost);
  cuda_error_handle();
}

void cuda_error_handle(){
  hipError_t errSync  = hipGetLastError();
  if (errSync != hipSuccess){
    char buffer[50];
    sprintf(buffer, "Sync kernel error (%i): %s\n", errSync, hipGetErrorString(errSync));
    throw CudaException(buffer);
  }
  hipError_t errAsync = hipDeviceSynchronize();  
  if (errAsync != hipSuccess){
    char buffer[50];
    sprintf(buffer, "Async kernel error (%i): %s\n", errAsync, hipGetErrorString(errAsync));
    throw CudaException(buffer);
  }
   
}




/**
 * Internal helpers
 **/
int blocksize(){ return CUDA_BLOCKSIZE; }
int nblocks(int n){ return (n + blocksize()-1) / blocksize(); }
